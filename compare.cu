
#include <hip/hip_runtime.h>
// Actually, there are no rounding errors due to results being accumulated in an arbitrary order..
// Therefore EPSILON = 0.0f is OK
#define EPSILON 0.0000000001f

extern "C" __global__ void compare(float *C, int *faultyElems, int iters) {
	int iterStep = blockDim.x*blockDim.y*gridDim.x*gridDim.y;
	int myIndex = (blockIdx.y*blockDim.y + threadIdx.y)* // Y
		gridDim.x*blockDim.x + // W
		blockIdx.x*blockDim.x + threadIdx.x; // X

	int myFaulty = 0;
	for (int i = 1; i < iters; ++i)
		if (fabsf(C[myIndex] - C[myIndex + i*iterStep]) > EPSILON)
			myFaulty++;

	atomicAdd(faultyElems, myFaulty);
}
